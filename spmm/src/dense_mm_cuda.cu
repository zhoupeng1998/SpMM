#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "timer.h"
#include "dense_mm_cuda.h"
#include "adj_matrix_dense.h"
#include "spmm_cuda.h"

__managed__ int numrows;

__global__ void dense_mm_kernel(INT* A_gpu, INT* B_gpu, INT* C_gpu) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i * SIZE < numrows; i++) {
        int i1 = i * SIZE + tid;
        if (i1 < numrows) {
            for (int j = 0; j < numrows; j++) {
                for (int k = 0; k < numrows; k++) {
                    C_gpu[i1 * numrows + j] += A_gpu[i1 * numrows + k] * B_gpu[k * numrows + j];
                }
            }
        }
    }
}

AdjMatrixDenseLinear dense_mm_cuda(AdjMatrixDenseLinear& A, AdjMatrixDenseLinear& B) {
    INT* A_gpu;
    INT* B_gpu;
    INT* C_gpu;
    INT* C_cpu;

    numrows = A.size();
    C_cpu = (INT*)malloc(A.size() * A.size() * sizeof(INT));

    hipMalloc(&A_gpu, A.size() * A.size() * sizeof(INT));
    hipMalloc(&B_gpu, B.size() * B.size() * sizeof(INT));
    hipMalloc(&C_gpu, A.size() * B.size() * sizeof(INT));

    hipMemset(C_gpu, 0, A.size() * A.size() * sizeof(INT));

    hipMemcpy(A_gpu, A.data(), A.size() * A.size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B.data(), B.size() * B.size() * sizeof(INT), hipMemcpyHostToDevice);

    // call kernel
    clock_start_cuda();
    dense_mm_kernel<<<GRIDSIZE, BLOCKSIZE>>>(A_gpu, B_gpu, C_gpu);
    clock_stop_cuda();

    hipMemcpy(C_cpu, C_gpu, A.size() * B.size() * sizeof(INT), hipMemcpyDeviceToHost);
    AdjMatrixDenseLinear C(A.size(), C_cpu);

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
    free(C_cpu);

    return C;
}