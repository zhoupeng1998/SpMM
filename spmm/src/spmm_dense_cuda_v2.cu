#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "spmm_cuda.h"
#include "data.h"
#include "adj_matrix_csr.h"
#include "adj_matrix_dense.h"

__managed__ int numrows;

__global__ void csr_spmm_dense_kernel_v2(INT* A_row, INT* A_col, INT* A_val, INT* B_row, INT* B_col, INT* B_val, INT* C_gpu) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i * SIZE < numrows; i++) {
        INT i1 = i * SIZE + tid;
        if (i1 < numrows) {
            for (int i2 = A_row[i1]; i2 < A_row[i1 + 1]; i2++) {
                INT colA = A_col[i2];
                for (int i3 = B_row[colA]; i3 < B_row[colA + 1]; i3++) {
                    INT colB = B_col[i3];
                    C_gpu[i1 * numrows + colB] += A_val[i2] * B_val[i3];
                }
            }
        }
    }
}

AdjMatrixDense csr_spmm_dense_cuda_v2(AdjMatrixCSR& A, AdjMatrixCSR& B) {
    INT* A_row;
    INT* A_col;
    INT* A_val;
    INT* B_row;
    INT* B_col;
    INT* B_val;

    INT* C_cpu;
    INT* C_gpu;

    numrows = A.num_rows();
    C_cpu = (INT*)malloc(numrows * numrows * sizeof(INT));

    hipMalloc(&A_row, (A.num_rows() + 1) * sizeof(INT));
    hipMalloc(&A_col, A.num_size() * sizeof(INT));
    hipMalloc(&A_val, A.num_size() * sizeof(INT));
    hipMalloc(&B_row, (B.num_rows() + 1) * sizeof(INT));
    hipMalloc(&B_col, B.num_size() * sizeof(INT));
    hipMalloc(&B_val, B.num_size() * sizeof(INT));

    hipMalloc(&C_gpu, numrows * numrows * sizeof(INT));
    hipMemset(C_gpu, 0, numrows * numrows * sizeof(INT));

    hipMemcpy(A_row, A.get_rows(), (A.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_col, A.get_cols(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_val, A.get_vals(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_row, B.get_rows(), (B.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_col, B.get_cols(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_val, B.get_vals(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);

    // call kernel
    csr_spmm_dense_kernel_v2<<<GRIDSIZE, BLOCKSIZE>>>(A_row, A_col, A_val, B_row, B_col, B_val, C_gpu);

    hipMemcpy(C_cpu, C_gpu, numrows * numrows * sizeof(INT), hipMemcpyDeviceToHost);
    AdjMatrixDense C(numrows, C_cpu);

    hipFree(A_row);
    hipFree(A_col);
    hipFree(A_val);
    hipFree(B_row);
    hipFree(B_col);
    hipFree(B_val);
    hipFree(C_gpu);
    free(C_cpu);

    return C;
}