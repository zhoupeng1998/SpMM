#include "hip/hip_runtime.h"
#include "data.h"
#include "adj_matrix_csr.h"
#include "spmm_cuda.h"
#include "stdio.h"
#include <iostream>


// #define GRIDSIZE 128
// #define BLOCKSIZE 1024
#define SIZE GRIDSIZE*BLOCKSIZE

__managed__ INT numrows;


__global__ void GetNNZ(INT* A_row, INT* A_col, INT* A_val, INT* B_row, INT* B_col, INT* B_val, INT* C_row, INT* work,INT rows) 
{
	const int laneId = threadIdx.x;
	const int warpId = blockIdx.x;
	
	INT* nonzeros;
	INT rowAStart, rowAEnd, rowBStart, rowBEnd;
	INT nnz;
	INT colC;
	
	extern __shared__ int nzCount[];
	
	nonzeros = &work[warpId * rows];
	
	// Iterate through each assigned row in A.
	for(INT rowA = warpId; rowA < rows; rowA += gridDim.x)
	{
    //printf("%ld, %d\n",rowA,gridDim.x);
		rowAStart = A_row[rowA];

    //printf("%ld, %d\n",A_row[3],gridDim.x);

		rowAEnd = A_row[rowA + 1];
		// There are no non-zeros in this row so continue
		if(rowAStart == rowAEnd)
		{
			if (laneId == 0)
				C_row[rowA] = 0;
			__syncthreads();
			continue;
		}

		// Reset the nz counts
		nzCount[laneId] = 0;
		
		// reset the nonzeros table
		for (INT i=laneId; i<rows; i+= warpSize)
		{
			nonzeros[i] = 0;
		}
		__syncthreads();
		
		for(INT i = rowAStart; i < rowAEnd; ++i)
		{
			rowBStart = B_row[A_col[i]];
			rowBEnd = B_row[A_col[i]+1];

			for (INT j = rowBStart + laneId; j < rowBEnd; j += warpSize)
			{
				colC = B_col[j];
				nzCount[laneId] += nonzeros[colC] == 0;
				nonzeros[colC] = 1;
			}
      
			__syncthreads();
		}

		if(laneId == 0)
		{
			nnz = nzCount[0];
			for(INT i = 1; i < BLOCKSIZE; ++i)
			{
				nnz += nzCount[i];
			}
			C_row[rowA] = nnz;

		}
		
		__syncthreads();
	}
}

__global__ void GetVals(INT* A_row, INT* A_col, INT* A_val, INT* B_row, INT* B_col, INT* B_val, 
INT* C_row, INT* C_col, INT* C_val, int* indexTable)
{
	const int laneId = threadIdx.x;
	const int bloackId = blockIdx.x;
	
	__shared__ unsigned int back;
	
	int rowAStart; // The index into A.jc and A.val
	int rowAEnd; // The boundary index for A
	float valA; // The value of the current A nonzero
	int rowBStart; // The index into B.jc and B.val
	int rowBEnd; // The boundary index for B
	int colB; // The current column in B being used
	int rowCStart; // The index into C.jc and C.val
	int rowCEnd; // The boundary index for C
	int hash; // The calculated hash value
	int i, j; // Loop iterators

	// Set the global hash table to point to the space
	// used by this warp
	int* gColHashTable;
	float* gValHashTable;
	int globalEntries;
	
	indexTable = &indexTable[C.cols * blockId];
	
	if(laneId == 0)
		back = 0;
	
	for(int rowA = blockId; rowA < numrows; rowA += gridDim.x)
	{
		rowAStart = A_row[rowA];
		rowAEnd = A_row[rowA + 1];
		for(i = laneId; i < numrows; ++i)
		{
			indexTable[i] = -1;
		}
		__syncthreads();

		// Set the location of the global hash table
		rowCStart = C_row[rowA];
		rowCEnd = C_row[rowA + 1];
		globalEntries = rowCEnd - rowCStart;
		gColHashTable = &C_col[rowCStart];
		gValHashTable = &C_val[rowCStart];
		for(i = rowAStart; i < rowAEnd; ++i)
		{
			valA = A_val[i];
			rowBStart = B_row[A_col[i]];
			rowBEnd = B_row[A_col[i] + 1];
			int curIdx;
			int* storeInt;
			float* storeFloat;
			float valB;
			for(j = rowBStart + laneId; __any(j < rowBEnd); j += warpSize)
			{
				colB = j < rowBEnd ? B_col[j] : -1;
				curIdx = colB == -1 ? -1 : indexTable[colB];
				hash = colB != -1 && curIdx == -1 ? atomicInc(&back, globalEntries - 1) : curIdx;
				storeInt = hash == -1 ? &hash : &indexTable[colB];
				*storeInt = hash;
				storeInt = hash == -1 ? &colB : &gColHashTable[hash];
				*storeInt = colB;
				valB = colB == -1 ? 1 : B_val[j];
				storeFloat = hash == -1 ? &valA : &gValHashTable[hash];
				*storeFloat += valB * valA;
			}
		} // For each nonzero in the A row
	} // For each assigned row in A
}




AdjMatrixCSR csr_spmm_cuda(AdjMatrixCSR& A, AdjMatrixCSR& B) {
    INT* A_row;
    INT* A_col;
    INT* A_val;
    INT* B_row;
    INT* B_col;
    INT* B_val;
    INT* C_row;
    INT* C_row_gpu;
	INT* C_col;
	INT* C_col_gpu;
	INT* C_val;
	INT* C_val_gpu;
    INT* work;
    
    numrows = A.num_rows();
    C_row = (INT*)malloc(sizeof(INT) * (numrows+1));

    hipMalloc(&A_row, (A.num_rows() + 1) * sizeof(INT));
    hipMalloc(&A_col, A.num_size() * sizeof(INT));
    hipMalloc(&A_val, A.num_size() * sizeof(INT));
    hipMalloc(&B_row, (B.num_rows() + 1) * sizeof(INT));
    hipMalloc(&B_col, B.num_size() * sizeof(INT));
    hipMalloc(&B_val, B.num_size() * sizeof(INT));
    hipMalloc(&C_row_gpu, (A.num_rows() + 1) * sizeof(INT));
    hipMalloc(&work, 1024*B.num_rows() * sizeof(INT));

    hipMemcpy(A_row, A.get_rows(), (A.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_col, A.get_cols(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_val, A.get_vals(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_row, B.get_rows(), (B.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_col, B.get_cols(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_val, B.get_vals(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);

    // call kernel

    GetNNZ<<<GRIDSIZE, BLOCKSIZE,numrows>>>(A_row, A_col, A_val, B_row, B_col, B_val, C_row_gpu, work,numrows);
    hipMemcpy(C_row, C_row_gpu, (A.num_rows() + 1) * sizeof(INT), hipMemcpyDeviceToHost);

    // prefix sum
    C_row[0] = 0;
    for (INT i = 0; i < numrows; i++) {
        C_row[i+1] += C_row[i];
    }

	hipMalloc(&C_col_gpu, C_row[numrows+1] * sizeof(INT));
	hipMalloc(&C_val_gpu, C_row[numrows+1] * sizeof(INT));
	


	GetVals<<<GRIDSIZE, BLOCKSIZE>>>(A_row, A_col, A_val, B_row, B_col, B_val, C_row_gpu,C_col_gpu,C_val_gpu, work,numrows);
	hipMemcpy(C_col, C_col_gpu, (C_row[numrows+1] * sizeof(INT), hipMemcpyDeviceToHost);
	hipMemcpy(C_val, C_val_gpu, (C_row[numrows+1] * sizeof(INT), hipMemcpyDeviceToHost);

    // hipMemcpy to host
    AdjMatrixCSR result(A.num_rows(), 0, C_row, NULL, NULL);
    
    result.size=C_row[numrows+1];
    return result;
}