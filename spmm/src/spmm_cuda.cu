#include "hip/hip_runtime.h"
#include "data.h"
#include "adj_matrix_csr.h"
#include "spmm_cuda.h"
#include "timer.h"

__managed__ int numrows;

__global__ void csr_spmm_symbolic(INT* A_row, INT* A_col, INT* A_val, INT* B_row, INT* B_col, INT* B_val, INT* C_row, INT* work) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //__shared__ INT work[BLOCKSIZE];
    for (int i = 0; i < numrows; i += SIZE) {
        INT i1 = i * SIZE + tid;
        if (i1 >= numrows) break;
        INT MARK = i1 + 1;
        INT count = 0;
        for (INT i2 = A_row[i1]; i2 < A_row[i1+1]; i2++) {
            INT j = A_col[i2];
            // assert(j >= 0 && j < numrows);
            for (INT i3 = B_row[j]; i3 < B_row[j+1]; i3++) {
                INT col = B_col[i3];
                // assert(col >= 0 && col < numrows);
                if (work[col] != MARK) {
                    count++;
                    work[col] = MARK;
                }
            }
        }
        C_row[i1+1] = count;
    }
    // prefix sum at host
}

AdjMatrixCSR csr_spmm_cuda(AdjMatrixCSR& A, AdjMatrixCSR& B) {
    INT* A_row;
    INT* A_col;
    INT* A_val;
    INT* B_row;
    INT* B_col;
    INT* B_val;
    INT* C_row;
    INT* C_row_gpu;
    INT* work;
    
    numrows = A.num_rows();
    C_row = (INT*)malloc(sizeof(INT) * (numrows+1));

    hipMalloc(&A_row, (A.num_rows() + 1) * sizeof(INT));
    hipMalloc(&A_col, A.num_size() * sizeof(INT));
    hipMalloc(&A_val, A.num_size() * sizeof(INT));
    hipMalloc(&B_row, (B.num_rows() + 1) * sizeof(INT));
    hipMalloc(&B_col, B.num_size() * sizeof(INT));
    hipMalloc(&B_val, B.num_size() * sizeof(INT));
    hipMalloc(&C_row_gpu, (A.num_rows() + 1) * sizeof(INT));
    hipMalloc(&work, B.num_rows() * sizeof(INT));

    hipMemcpy(A_row, A.get_rows(), (A.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_col, A.get_cols(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(A_val, A.get_vals(), A.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_row, B.get_rows(), (B.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_col, B.get_cols(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);
    hipMemcpy(B_val, B.get_vals(), B.num_size() * sizeof(INT), hipMemcpyHostToDevice);

    // call kernel
    csr_spmm_symbolic<<<GRIDSIZE, BLOCKSIZE>>>(A_row, A_col, A_val, B_row, B_col, B_val, C_row, work);
    hipMemcpy(C_row, C_row_gpu, (A.num_rows() + 1) * sizeof(INT), hipMemcpyHostToDevice);
    // prefix sum
    C_row[0] = 0;
    for (INT i = 0; i < numrows; i++) {
        C_row[i+1] += C_row[i];
    }
    // TODO: implement numeric

    // hipMemcpy to host
    AdjMatrixCSR result(A.num_rows(), 0, C_row, NULL, NULL);
    return result;
}